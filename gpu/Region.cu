#include "hip/hip_runtime.h"
#include "Region.h"
#include <hip/hip_runtime.h>

// our kernel for edge updates
// parameters:
// g: graph
// epsilon: epsilon
// numThreadUpdates: number of updates in each thread
// lambdaGlobal: global lambda array
// runFlag: a flag that controls when we want to terminate the array
// rangeRandNums: random numbers (defined by the graph)
template<typename T, typename S>
__global__ void EdgeUpdateKernel(MPGraph<T, S>* g, T epsilon, size_t* numThreadUpdates, T* lambdaGlobal, volatile int* runFlag, int numThreads)
{
     int tx = threadIdx.x + blockIdx.x * blockDim.x;

     if(tx < numThreads)
     {
         int uid;
         hiprandState_t state;
         hiprand_init(clock64(),tx,0,&state);

         // allocate space for edge workspace
         typename MPGraph<T, S>::REdgeWorkspaceID rew;
         rew = g->AllocateReparameterizeEdgeWorkspaceMem(epsilon);

         // allocate an array that will act as our base
         size_t msgSize = g->GetLambdaSize();
         T* devLambdaBase = (T*)malloc(msgSize * sizeof(T));
         memset(devLambdaBase, T(0), sizeof(T) * msgSize);

         int rangeRandNums = g->NumberOfEdges();




         //for(int i = 0; i < 200; i++)
         //{
         	uid = floorf(hiprand_uniform(&state) * rangeRandNums);
		g->CopyMessagesForEdge(lambdaGlobal, devLambdaBase, uid);
		g->ReparameterizeEdge(devLambdaBase, uid, epsilon, false, rew);
		g->UpdateEdge(devLambdaBase, lambdaGlobal, uid, false);
//`
		numThreadUpdates[tx]++;
		__syncthreads();
         	
	//}
//
//         // free device pointers
         g->DeAllocateReparameterizeEdgeWorkspaceMem(rew);
         free(devLambdaBase);
//
         //atomicAdd(runFlag, numThreads);
//
     }
//
}


template<typename T, typename S>
__global__ void RegionUpdateKernel(MPGraph<T, S>* g, T epsilon, size_t* numThreadUpdates, T* lambdaGlobal, volatile int* runFlag, int numThreads)
{
     int tx = threadIdx.x + blockIdx.x * blockDim.x;

     if(tx < numThreads)
     {
         int uid;
         hiprandState_t state;
         hiprand_init(clock64(),tx,0,&state);

         // allocate space for edge workspace
         typename MPGraph<T, S>::RRegionWorkspaceID rew;
         rew = g->AllocateReparameterizeRegionWorkspaceMem(epsilon);

         // allocate an array that will act as our base
         size_t msgSize = g->GetLambdaSize();
         T* devLambdaBase = (T*)malloc(msgSize * sizeof(T));
         memset(devLambdaBase, T(0), sizeof(T) * msgSize);

         int rangeRandNums = g->NumberOfRegionsWithParents();




         for(int i = 0; i < 500; i++)
         {
         	uid = floorf(hiprand_uniform(&state) * rangeRandNums);
		g->CopyMessagesForStar(lambdaGlobal, devLambdaBase, uid);
		g->ReparameterizeRegion(devLambdaBase, uid, epsilon, false, rew);
		g->UpdateRegion(devLambdaBase, lambdaGlobal, uid, false);
//`
		numThreadUpdates[tx]++;
		__syncthreads();
         	
	}
//
//         // free device pointers
         g->DeAllocateReparameterizeRegionWorkspaceMem(rew);
         free(devLambdaBase);
//
         //atomicAdd(runFlag, numThreads);
//
     }
//
}


template<typename T, typename S>
int CudaAsyncRMPThread<T,S>::CudaRunMP(MPGraph<T, S>& g, T epsilon, int numIterations, int numThreads, int WaitTimeInMS) {

    size_t msgSize = g.HostGetLambdaSize();

    std::cout << "Num threads " << numThreads << std::endl;

    // handle this case later.i
    if (msgSize == 0) {
        typename MPGraph<T, S>::DualWorkspaceID dw = g.HostAllocateDualWorkspaceMem(epsilon);
        std::cout << "0: " << g.HostComputeDual(NULL, epsilon, dw) << std::endl;
        g.HostDeAllocateDualWorkspaceMem(dw);
        return 0;
    }
    std::cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1);

    // allocate device pointers for lambda global
    T* devLambdaGlobal = NULL;
    gpuErrchk(hipMalloc((void**)&devLambdaGlobal, sizeof(T) * msgSize));
    gpuErrchk(hipMemset((void*)devLambdaGlobal, T(0), sizeof(T)*msgSize));


    // allocate on host memory for cuda streaming
    T* lambdaGlob = NULL;
    gpuErrchk(hipHostMalloc((void**)&lambdaGlob, sizeof(T)*msgSize));
    gpuErrchk(hipMemset((void*)lambdaGlob, T(0), sizeof(T)*msgSize));


    

    // allocate space and copy graph to GPU
    MPGraph<T,S>* gPtr = NULL;
    gpuErrchk(hipMalloc((void**)&gPtr, sizeof(g)));
    gpuErrchk(hipMemcpy(gPtr, &g, sizeof(g), hipMemcpyHostToDevice));

    // initialize the number of region updates
    size_t* numThreadUpdates = NULL;
    size_t* hostThreadUpdates = new size_t[numThreads];
    gpuErrchk(hipMalloc((void**)&numThreadUpdates, numThreads * sizeof(size_t)));
    gpuErrchk(hipMemset((void*)numThreadUpdates, 0, numThreads * sizeof(size_t)));

    // allocate run flag
    int* devRunFlag = NULL;
    gpuErrchk(hipMalloc((void**)&devRunFlag, sizeof(int)));
    gpuErrchk(hipMemset((void*)devRunFlag, 0, sizeof(int)));

    // create an asynchronous cuda stream
    // we only have two streams, the main (CPU) stream, and the GPU one
    // CPU stream only copies back every so often (or writes to the GPU)
    // GPU is executing
    hipStream_t streamCopy, streamExec;
    gpuErrchk(hipStreamCreate(&streamCopy));
    gpuErrchk(hipStreamCreate(&streamExec));


    // create a ThreadSync object (not necessary at all, but hey, I wanna
    // make sure this actually works)
    ThreadSync<T, S> sy(numThreads, lambdaGlob, epsilon, &g);

    // grid/block dimensions
    dim3 DimGrid(ceil(numThreads * 1.0 / BLOCK_SIZE),1,1);
    dim3 DimBlock(BLOCK_SIZE,1,1);
    int stopFlag = 1;

    std::cout << "Executing kernel..." << std::endl;


    // start the kernel
   // EdgeUpdateKernel<<<DimGrid, DimBlock, 0, streamExec>>>(gPtr, epsilon, numThreadUpdates, devLambdaGlobal, devRunFlag, numThreads);

    RegionUpdateKernel<<<DimGrid, DimBlock, 0, streamExec>>>(gPtr, epsilon, numThreadUpdates, devLambdaGlobal, devRunFlag, numThreads);
    /*    
    for (int k = 0; k < numIterations; ++k)
    {
        std::cout << "Iteration " << k << std::endl;
        std::this_thread::sleep_for(std::chrono::milliseconds(WaitTimeInMS));

        hipMemcpyAsync(lambdaGlob, devLambdaGlobal, sizeof(T)*msgSize, hipMemcpyDeviceToHost, streamCopy);
        sy.ComputeDualNoSync();

    }
*/
   
    gpuErrchk(hipMemcpyAsync(devRunFlag, &stopFlag, sizeof(int), hipMemcpyHostToDevice, streamCopy));

    hipDeviceSynchronize();
    // now, we can block
    gpuErrchk(hipMemcpy(hostThreadUpdates, numThreadUpdates, sizeof(size_t)*numThreads, hipMemcpyDeviceToHost));

    g.ResetMessageMemory();

    hipMemcpy(lambdaGlob, devLambdaGlobal, sizeof(T)*msgSize, hipMemcpyDeviceToHost);
    sy.ComputeDualNoSync();
    gpuErrchk(hipMemcpy(&stopFlag, devRunFlag, sizeof(int), hipMemcpyDeviceToHost));
    if(stopFlag == 1)
    {
        std::cout << "Kernel Terminated" << std::endl;
    }

    size_t regionUpdates = 0;
   for(int k=0;k<numThreads;++k) {
        size_t tmp = hostThreadUpdates[k];
       // std::cout << "Thread " << k << ": " << tmp << std::endl;
        regionUpdates += tmp;
   }

    hipFree(gPtr);
    hipHostFree(lambdaGlob);
    hipFree(devRunFlag);
    hipFree(devLambdaGlobal);
    hipHostFree(lambdaGlob);
    delete [] hostThreadUpdates;
    hipStreamDestroy(streamCopy);
    hipStreamDestroy(streamExec);

    hipDeviceReset();

    std::cout << "Region updates: " << regionUpdates << std::endl;
    std::cout << "Total regions:  " << g.HostNumberOfRegionsWithParents() << std::endl;

//    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
    std::cout << "Terminating program." << std::endl;
    return 0;
}
