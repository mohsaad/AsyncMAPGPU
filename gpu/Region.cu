#include "hip/hip_runtime.h"
#include "Region.h"
#include <hip/hip_runtime.h>

// our kernel for edge updates
// parameters:
// g: graph
// epsilon: epsilon
// numThreadUpdates: number of updates in each thread
// lambdaGlobal: global lambda array
// runFlag: a flag that controls when we want to terminate the array
// rangeRandNums: random numbers (defined by the graph)
template<typename T, typename S>
__global__ void EdgeUpdateKernel(MPGraph<T, S>* g, T epsilon, size_t* numThreadUpdates, T* lambdaGlobal, volatile int* runFlag, int numThreads)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if(tx < numThreads)
    {
        int uid;
        hiprandState_t state;
        hiprand_init(clock64(),tx,0,&state);

        // allocate space for edge workspace
        typename MPGraph<T, S>::REdgeWorkspaceID rew;
        rew = g->AllocateReparameterizeEdgeWorkspaceMem(epsilon);

        // allocate an array that will act as our base
        size_t msgSize = g->GetLambdaSize();
        T* devLambdaBase = (T*)malloc(msgSize * sizeof(T));
        //memset(devLambdaBase, T(0), sizeof(T) * msgSize);

        int rangeRandNums = g->NumberOfEdges();


        uid = floorf(hiprand_uniform(&state) * rangeRandNums);
	    g->CopyMessagesForEdge(lambdaGlobal, devLambdaBase, uid);
	    g->ReparameterizeEdge(devLambdaBase, uid, epsilon, false, rew);
	    g->UpdateEdge(devLambdaBase, lambdaGlobal, uid, false);

        g->DeAllocateReparameterizeEdgeWorkspaceMem(rew);
        free(devLambdaBase);
    }
}

__device__ bool checkFlag(volatile bool* flag)
{
    return *flag;
}


template<typename T, typename S>
__global__ void RegionUpdateKernel(MPGraph<T, S>* g, T epsilon, size_t* numThreadUpdates, T* lambdaGlobal, T* lambdaBase, volatile bool* runFlag, int numThreads)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if(tx < numThreads)
    {
        int uid;
        hiprandState_t state;
        hiprand_init(clock64(),tx,0,&state);

        // allocate space for edge workspace
        typename MPGraph<T, S>::RRegionWorkspaceID rew;
        rew = g->AllocateReparameterizeRegionWorkspaceMem(epsilon);

        // allocate an array that will act as our base
        size_t msgSize = g->GetLambdaSize();
	    T* devLambdaBase = (T*)malloc(msgSize * sizeof(T));
	    memset(devLambdaBase, T(0), sizeof(T) * msgSize);

        int rangeRandNums = g->NumberOfRegionsWithParents();

        uid = floorf(hiprand_uniform(&state) * rangeRandNums);
	    g->CopyMessagesForStar(lambdaGlobal, devLambdaBase, uid);
	    g->ReparameterizeRegion(devLambdaBase, uid, epsilon, false, rew);
	    g->UpdateRegion(devLambdaBase, lambdaGlobal, uid, false);

    
        // free device pointers
        g->DeAllocateReparameterizeRegionWorkspaceMem(rew);
        free(devLambdaBase);


     }
}


template<typename T, typename S>
int CudaAsyncRMPThread<T,S>::CudaRunMP(MPGraph<T, S>& g, T epsilon, int numIterations, int numThreads, int WaitTimeInMS) {

    size_t msgSize = g.HostGetLambdaSize();

    std::cout << "Num threads " << numThreads << std::endl;

    // handle this case later.i
    if (msgSize == 0) {
        typename MPGraph<T, S>::DualWorkspaceID dw = g.HostAllocateDualWorkspaceMem(epsilon);
        std::cout << "0: " << g.HostComputeDual(NULL, epsilon, dw) << std::endl;
        g.HostDeAllocateDualWorkspaceMem(dw);
        return 0;
    }
    std::cout << std::setprecision(15);

    // allocate device pointers for lambda global
    T* devLambdaGlobal = NULL;
    gpuErrchk(hipMalloc((void**)&devLambdaGlobal, sizeof(T) * msgSize));
    gpuErrchk(hipMemset((void*)devLambdaGlobal, T(0), sizeof(T)*msgSize));


    // allocate on host memory for cuda streaming
    T* lambdaGlob = NULL;
    gpuErrchk(hipHostMalloc((void**)&lambdaGlob, sizeof(T)*msgSize));
    gpuErrchk(hipMemset((void*)lambdaGlob, T(0), sizeof(T)*msgSize));




    // allocate space and copy graph to GPU
    MPGraph<T,S>* gPtr = NULL;
    gpuErrchk(hipMalloc((void**)&gPtr, sizeof(g)));
    gpuErrchk(hipMemcpy(gPtr, &g, sizeof(g), hipMemcpyHostToDevice));

    // initialize the number of region updates
    size_t* numThreadUpdates = NULL;
    size_t* hostThreadUpdates = new size_t[numThreads];
    gpuErrchk(hipMalloc((void**)&numThreadUpdates, numThreads * sizeof(size_t)));
    gpuErrchk(hipMemset((void*)numThreadUpdates, 0, numThreads * sizeof(size_t)));


    // allocate all the base lambdas
    T* indivLambda;
    gpuErrchk(hipMalloc((void**)&indivLambda, sizeof(T)*msgSize*numThreads));
    gpuErrchk(hipMemset((void*)indivLambda, 0, sizeof(T)*msgSize*numThreads)); 

    // allocate run flag
    bool* devRunFlag = NULL;
    bool tmpTest = true;
    gpuErrchk(hipMalloc((void**)&devRunFlag, sizeof(bool)));
    gpuErrchk(hipMemcpy(devRunFlag, &tmpTest, sizeof(bool), hipMemcpyHostToDevice));

    // create an asynchronous cuda stream
    // we only have two streams, the main (CPU) stream, and the GPU one
    // CPU stream only copies back every so often (or writes to the GPU)
    // GPU is executing
    hipStream_t streamCopy, streamExec;
    gpuErrchk(hipStreamCreate(&streamCopy));
    gpuErrchk(hipStreamCreate(&streamExec));


    // create a ThreadSync object (not necessary at all, but hey, I wanna
    // make sure this actually works)
    ThreadSync<T, S> sy(numThreads, lambdaGlob, epsilon, &g);

    // grid/block dimensions
    dim3 DimGrid(ceil(numThreads * 1.0 / BLOCK_SIZE),1,1);
    dim3 DimBlock(BLOCK_SIZE,1,1);
    bool stopFlag = false;

    std::cout << "Executing kernel..." << std::endl;



    RegionUpdateKernel<<<DimGrid, DimBlock>>>(gPtr, epsilon, numThreadUpdates, devLambdaGlobal, indivLambda, devRunFlag, numThreads);

    gpuErrchk(hipMemcpy(hostThreadUpdates, numThreadUpdates, sizeof(size_t)*numThreads, hipMemcpyDeviceToHost));


    hipMemcpy(lambdaGlob, devLambdaGlobal, sizeof(T)*msgSize, hipMemcpyDeviceToHost);
    sy.ComputeDualNoSync();
    std::cout << "Kernel Terminated" << std::endl;

    size_t regionUpdates = 0;


    //hipFree(gPtr);
    hipHostFree(lambdaGlob);
    //hipFree(devRunFlag);
    //hipFree(indivLambda);
    //hipFree(devLambdaGlobal);
    //hipHostFree(lambdaGlob);
    delete [] hostThreadUpdates;
    hipStreamDestroy(streamCopy);
    hipStreamDestroy(streamExec);

    hipDeviceReset();

    std::cout << "Region updates: " << regionUpdates << std::endl;
    std::cout << "Total regions:  " << g.HostNumberOfRegionsWithParents() << std::endl;

//    std::this_thread::sleep_for(std::chrono::milliseconds(1000));
    std::cout << "Terminating program." << std::endl;
    return 0;
}
